#include "hip/hip_runtime.h"
// Copyright 2018 Delft University of Technology
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
#include "../utils/Timer.hpp"
#include <iostream>
#include <cmath>
#include "../baseline/imgproc.hpp"
#include "../baseline/water.hpp"
#include "../utils/Histogram.hpp"
#include "water_cuda.hpp"
#include "imgproc_cuda.hpp"
std::shared_ptr<Image> runEnhanceStage(const Image *previous, const Histogram *hist, const WaterEffectOptions *options, Timer ts) {
  // Create a new image to store the result
  auto img_enhanced = std::make_shared<Image>(previous->width, previous->height);
  
  int numPixels = previous->height * previous->width;
  int numBlocks = (numPixels + 32 - 1) / 32;
  int blockSize = 32;

  // Move src image to device memory
  size_t img_size = sizeof(unsigned char) * numPixels * 4;
  unsigned char *src;
  hipMallocManaged(&src, img_size);
  hipMemcpy((void *)src, (void *)(previous->raw.data()), img_size, hipMemcpyHostToDevice);

  ts.start();

  // Determine the threshold from the histogram, by taking 10% of the maximum value in the histogram.
  auto max_hist = (int) (hist->max(0) * 0.1);

  unsigned char begin = 0;
  unsigned char end = 255;

  // Enhance each (non-alpha) channel of the source image
  for (int i = 0; i < 3; i++)
  {
    // Obtain the first intensity that is above the threshold.
    for (begin = 0; begin < hist->range; begin++) {
      if (hist->count(begin, i) > max_hist) {
        break;
      }
    }

    // Obtain the last intensity that is above the threshold.
    for (end = 255; end > begin; end--) {
      if (hist->count(end, i) > max_hist) {
        break;
      }
    }
    enhanceContrastLinearlyCuda<<<numBlocks, blockSize>>>(src, src, begin, end, i, numPixels);
    // Wait for completion
    hipDeviceSynchronize();
    hipGetLastError();
  }
  ts.stop();

  std::cout << "Stage: Contrast enhance:        " << ts.seconds() << " s." << std::endl;

  // Transfer enhanced image from device to host memory
  hipMemcpy(img_enhanced->raw.data(), (void *)src, img_size, hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(src);

  // Save the resulting image
  if (options->save_intermediate)
    img_enhanced->toPNG("output/" + options->img_name + "_enhancedCUDA.png");

  // Create and save the enhanced histogram (if enabled).
  if (options->enhance_hist) {
    auto enhanced_hist = getHistogram(img_enhanced.get());
    auto enhanced_hist_img = enhanced_hist.toImage();
    enhanced_hist_img->toPNG("output/" + options->img_name + "_enhanced_histogramCUDA.png");
  }

  return img_enhanced;
}
std::shared_ptr<Histogram> runHistogramStage(const Image *previous, const WaterEffectOptions *options, Timer ts) {
    // Histogram to hold result
    // Histogram *hist_res = new Histogram();
    auto hist_res = std::make_shared<Histogram>();

    // Determine # of threads to allocate
    int numPixels = previous->height * previous->width;
    int numBlocks = (numPixels + 32 - 1) / 32;
    int blockSize = 32;

    // Histogram stage
    
    if (options->histogram) {
   	 
	// Move src 
    size_t img_size = sizeof(unsigned char) * numPixels * 4;
    unsigned char *src;
    hipMallocManaged(&src, img_size);
    //move to device
    hipMemcpy((void *)src, (void *)(previous->raw.data()), img_size, hipMemcpyHostToDevice);

    // Set up device memory for histogram
    int *hist;
    size_t hist_size = sizeof(int) * 4 * 256;
    hipMallocManaged(&hist, hist_size);
    hipMemset(hist, 0, hist_size);

    ts.start();
    getHistogramCuda<<<numBlocks, blockSize>>>(src, numPixels, hist);
    // Wait for completion
    hipDeviceSynchronize();
    ts.stop();
    std::cout << "Stage: Histogram:        " << ts.seconds() << " s." << std::endl;

    // Copy the result data back to host
    hipMemcpy(hist_res->values.data(), hist, hist_size, hipMemcpyDeviceToHost);

    if (options->save_intermediate) {
      // Copy raw data into histogram object
      auto hist_img = hist_res->toImage();
      hist_img->toPNG("output/" + options->img_name + "_histogramCUDA.png");
    }
    hipFree(src);
    hipFree(hist); 
  }
  return hist_res;
}

std::shared_ptr<Image> runWaterEffectCUDA(const Image *src, const WaterEffectOptions *options) {
  /* REPLACE THIS CODE WITH YOUR OWN WATER EFFECT PIPELINE */
Timer ts;
  // Smart pointers to intermediate images:
  std::shared_ptr<Histogram> hist;
  std::shared_ptr<Image> img_result;
  // Histogram stage
  if (options->histogram)
  {
    hist = runHistogramStage(src, options, ts);
  }
  if (options->enhance)
  {
    if (hist == nullptr) {
      throw std::runtime_error("Cannot run enhance stage without histogram.");
    }
    img_result = runEnhanceStage(src, hist.get(), options, ts);
  }
  return nullptr;
  /* REPLACE THIS CODE WITH YOUR OWN WATER EFFECT PIPELINE */
}
